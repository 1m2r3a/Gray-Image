#include<stdio.h>
#include<stdlib.h>
#include <stdint.h>
#include "hip/hip_runtime.h"

#include <time.h>
#include <iomanip>
#include <iostream>

using namespace std;

struct BITMAPFILEHEADER{
   uint8_t type[2];
   uint32_t size;
   uint16_t reserved1;
   uint16_t reserved2;
   uint32_t offsetbits;
} __attribute__ ((packed));



struct BITMAPINFOHEADER{
   uint32_t size;
   uint32_t width;
   uint32_t height;
   uint16_t planes;
   uint16_t bitcount;
   uint32_t compression;
   uint32_t sizeimage;
   long xpelspermeter;
   long ypelspermeter;
   uint32_t colorsused;
   uint32_t colorsimportant;
} __attribute__ ((packed));



struct myRGB{
   uint8_t blue; 
   uint8_t green;
   uint8_t red;  
};
struct grayStruct{
   uint8_t B;
   uint8_t G;
   uint8_t R;
};

__global__ void kernel(myRGB* input, grayStruct* gray,int w,int h, int chunksize)
{  
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int totalsize = w*h;
	if( idx < totalsize)
   for( int i = idx*chunksize ; i< idx*chunksize+chunksize && i < totalsize; i++)
     {
      int graypixel = (int)(0.33* input[i].red + 0.33* input[i].green + 0.33* input[i].blue);
      gray[i].R = graypixel;gray[i].G = graypixel;gray[i].B = graypixel;
     }

}
int main(int argc, char* argv[])
{
  clock_t start = clock();

  BITMAPFILEHEADER source_head;
  BITMAPINFOHEADER source_info;

  FILE *in;
  FILE *out;

  if(!(in=fopen(argv[1],"rb")))
    {
    	printf("\ncan not open file");
    	exit(-1);
  }

  out=fopen("out_cuda.bmp","wb");

  fread(&source_head,sizeof(struct BITMAPFILEHEADER),1,in);
  fread(&source_info,sizeof(struct BITMAPINFOHEADER),1,in);
 
  int total_pixel = source_info.width * source_info.height;
  
  myRGB* h_pixel;
  h_pixel = (myRGB*) malloc (sizeof(myRGB)*total_pixel);
  fread(h_pixel,sizeof(myRGB),total_pixel,in);
  
  myRGB* d_pixel;
  hipMalloc((void **)&d_pixel, total_pixel*sizeof(myRGB));
  hipMemcpy(d_pixel, h_pixel,total_pixel*sizeof(myRGB), hipMemcpyHostToDevice);

  grayStruct* d_gray;
  hipMalloc( (void**) &d_gray, total_pixel*sizeof(grayStruct));
  hipMemset(d_gray,255,total_pixel*sizeof(grayStruct));


  const int BLOCK_SIZE = atoi(argv[2]);
  const int THREAD_SIZE = atoi(argv[3]);
  int chunksize=  ceil( (double)total_pixel/(BLOCK_SIZE*THREAD_SIZE));
  
  cout<< "Chunk Size: " << chunksize<< endl;
  kernel <<<BLOCK_SIZE ,THREAD_SIZE >>> (d_pixel, d_gray, source_info.width, source_info.height, chunksize);


  fwrite(&source_head,sizeof(struct BITMAPFILEHEADER),1,out);
  fwrite(&source_info,sizeof(struct BITMAPINFOHEADER),1,out);
 
  hipDeviceSynchronize();
  grayStruct* h_gray = (grayStruct*) malloc (total_pixel*sizeof(grayStruct));
 hipMemcpy(h_gray, d_gray, total_pixel*sizeof(grayStruct), hipMemcpyDeviceToHost);

  fwrite(h_gray,sizeof(grayStruct),total_pixel,out);

  fclose(in);
  fclose(out);
  
  clock_t end = clock();
  double t_time = (end - start)/(double)CLOCKS_PER_SEC;
  cout << "Time: " << std::setprecision(9) << t_time << endl;
  
  return 0;
}
